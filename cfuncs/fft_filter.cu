#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "fft_filter.h"

#define CHECK_CUDA(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", \
            __FILE__, __LINE__, hipGetErrorString(err)); \
        return -1; \
    } \
} while(0)

#define CHECK_CUFFT(call) do { \
    hipfftResult r = (call); \
    if (r != HIPFFT_SUCCESS) { \
        fprintf(stderr, "CUFFT error %s:%d: %d\n", \
            __FILE__, __LINE__, r); \
        return -1; \
    } \
} while(0)

// Kernel: zero out frequencies outside cutoff
__global__ void applyFilter(hipfftDoubleComplex *data, int N, int cutoff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        int freq = (i <= N/2) ? i : (i - N);
        if (abs(freq) > cutoff) {
            data[i].x = 0.0;
            data[i].y = 0.0;
        }
    }
}

// Initialize
int fft_filter_init(FFTFilterCtx *ctx, int N, int cutoff) {
    ctx->N = N;
    ctx->cutoff = cutoff;

    // Allocate device buffer
    CHECK_CUDA(hipMalloc(&ctx->d_data, N * sizeof(hipfftDoubleComplex)));

    // Create FFT plan
    hipfftHandle *plan = new hipfftHandle;
    CHECK_CUFFT(hipfftPlan1d(plan, N, HIPFFT_Z2Z, 1));
    ctx->plan = plan;

    return 0;
}

// Run one FFT-filter-IFFT
int fft_filter_run(FFTFilterCtx *ctx, double _Complex *arr) {
    int N = ctx->N;
    int cutoff = ctx->cutoff;
    hipfftHandle *plan = (hipfftHandle*)ctx->plan;
    hipfftDoubleComplex *d_data = (hipfftDoubleComplex*)ctx->d_data;

    // Copy in
    CHECK_CUDA(hipMemcpy(d_data, arr, N * sizeof(hipfftDoubleComplex),
                          hipMemcpyHostToDevice));

    // Forward FFT
    CHECK_CUFFT(hipfftExecZ2Z(*plan, d_data, d_data, HIPFFT_FORWARD));

    // Filter
    int block = 256;
    int grid = (N + block - 1) / block;
    applyFilter<<<grid, block>>>(d_data, N, cutoff);
    CHECK_CUDA(hipDeviceSynchronize());

    // Inverse FFT
    CHECK_CUFFT(hipfftExecZ2Z(*plan, d_data, d_data, HIPFFT_BACKWARD));

    // Copy back
    CHECK_CUDA(hipMemcpy(arr, d_data, N * sizeof(hipfftDoubleComplex),
                          hipMemcpyDeviceToHost));

    // Normalize
    for (int i = 0; i < N; i++) {
        arr[i] /= N;
    }

    return 0;
}

// Cleanup
void fft_filter_destroy(FFTFilterCtx *ctx) {
    if (ctx->plan) {
        hipfftHandle *plan = (hipfftHandle*)ctx->plan;
        hipfftDestroy(*plan);
        delete plan;
        ctx->plan = NULL;
    }
    if (ctx->d_data) {
        hipFree(ctx->d_data);
        ctx->d_data = NULL;
    }
}
